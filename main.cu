////
//// Created by harish on 17.07.20.
////
//
//// Begin of Thrust libraries
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/remove.h>
//#include <thrust/copy.h>
//#include <thrust/execution_policy.h>
//#include <thrust/find.h>
//// End of Thrust libraries
//
#include <chrono>
using namespace std::chrono;
//
//// Begin of Arrayfire libraries
//#include <arrayfire.h>
//#include <af/array.h>
//// End of Arrayfire libraries
//
////#include <cudf.h> //installed using command "sudo apt-get install libcudf-dev"
//
//// Begin of STL libraries
#include <iostream>
#include <fstream>
#include <sstream>
#include<algorithm>
#include <array>

//// End of STL libraries
//
//#include "arrayfireops.h"
//
//***************************************************************************************************//
//*************** READ TABLE DATA FROM .TBL FILES ***************************************************//
//***************************************************************************************************//
std::vector<std::vector<int>> getLineItemData() {
    std::string filename = "/home/harish/Documents/lineitem/lineitem.tbl";

    std::vector<std::vector<int>> lineitemData;

    std::vector<int> rowVector;

    std::ifstream lineitemFile(filename);

/*    if(!lineitemFile.is_open())
        throw std::runtime_error("could not open file");*/

    std::string row;
    std::string values;

    while (lineitemFile.good()) {
        std::getline(lineitemFile, row);
        std::stringstream ss(row);
        int count = 0;
        while (std::getline(ss,values,'|')){
            if(count < 8 or (count >=10 and count <=12)){
                if(count >= 4 and count <=7) {
                    int val = std::stof(values) * 100;
                    rowVector.push_back(val);
                }
                else if(count >=10 and count <=12){
                    values.erase(remove(values.begin(), values.end(), '-'), values.end());
                    rowVector.push_back(std::stoi(values));
                }
                else{
                    rowVector.push_back(std::stoi(values));
                }
            }
            count++;
        }
        lineitemData.push_back(rowVector);
        rowVector.clear();
    }
    return lineitemData;
}

//***************************************************************************************************//
//*************** TRANSPOSE TABLE DATA FOR COLUMN FORMAT TO LOAD INTO DEVICE ************************//
//***************************************************************************************************//
std::vector<std::vector<int>> getTransposedVector(const std::vector<std::vector<int>> &lineitemData) {
    std::vector<std::vector<int>> transposedVec(lineitemData[0].size(),
                                                std::vector<int>(lineitemData.size()));
    for (size_t i = 0; i < lineitemData.size(); ++i)
        for (size_t j = 0; j < lineitemData[0].size(); ++j)
            transposedVec[j][i] = lineitemData[i][j];
    return transposedVec;
}
//
////***************************************************************************************************//
////*************** ARRAYFIRE - HOW TO SET DIFFERENT BACKENDS *****************************************//
////***************************************************************************************************//
//
//void testBackend()
//{
//    af::info();
//    af_print(af::randu(5, 4));
//}
//
//void testArrayFire()
//{
//    try {
//        printf("Trying CPU Backend\n");
//        af::setBackend(AF_BACKEND_CPU);
//        testBackend();
//    } catch (af::exception& e) {
//        printf("Caught exception when trying CPU backend\n");
//        fprintf(stderr, "%s\n", e.what());
//    }
//
//    try {
//        printf("Trying CUDA Backend\n");
//        af::setBackend(AF_BACKEND_CUDA);
//        testBackend();
//    } catch (af::exception& e) {
//        printf("Caught exception when trying CUDA backend\n");
//        fprintf(stderr, "%s\n", e.what());
//    }
//
//    try {
//        printf("Trying OpenCL Backend\n");
//        af::setBackend(AF_BACKEND_OPENCL);
//        testBackend();
//    } catch (af::exception& e) {
//        printf("Caught exception when trying OpenCL backend\n");
//        fprintf(stderr, "%s\n", e.what());
//    }
//
//
//    // Create an array on the host, copy it into an ArrayFire 2x3 ArrayFire
//    // array
//    float host_ptr[] = {0, 1, 2, 3, 4, 5};
//    af::array a(2, 3, host_ptr);
//
//    // Create a CUDA device pointer, populate it with data from the host
//    float *device_ptr;
//    hipMalloc((void **)&device_ptr, 6 * sizeof(float));
//    hipMemcpy(device_ptr, host_ptr, 6 * sizeof(float), hipMemcpyHostToDevice);
//
//    // Convert the CUDA-allocated device memory into an ArrayFire array:
//    af::array b(2, 3, device_ptr, afDevice);  // Note: afDevice (default: afHost)
//    // Note that ArrayFire takes ownership over `device_ptr`, so memory will
//    // be freed when `b` id destructed. Do not call hipFree(device_ptr)!
//}
//
////***************************************************************************************************//
////*************** THRUST - FILTER OPERATION *********************************************************//
////***************************************************************************************************//
//
//void filter_thrust() {// this following check confirmed addition of --expt-extended-lambda in CmakeLists.txt
//// is necessary to executed lambda functions
//#ifndef __CUDACC_EXTENDED_LAMBDA__
//    std::cout << "compile with extended lambdas " << std::endl;
//#endif
//
//    // get the table content as 2D vector
//    std::vector<std::vector<int>> lineitemData;
//    lineitemData = getLineItemData();
//
//    // transpose the table data to read in column format
//    std::vector<std::vector<int>> transposedVec = getTransposedVector(lineitemData);
//
//    // the table data in column format is host vector H
//    thrust::host_vector<std::vector<int>> H(transposedVec);
//
//    std::cout << "H has size " << H.size() << std::endl;
//    std::cout << "H has size " << H[0].size() << std::endl;
//
//    // the predicate column data is transferred/copied to the device vector
//    thrust::device_vector<int> D0 = H[8];
//    std::cout << "D has size before filtering " << D0.size() << std::endl;
//
//    // Remove_If operation from thrust on device vector
///*    D0.erase(thrust::remove_if(D0.begin(), D0.end(), [=] __device__(const int x){
//        return !(x >= 19940101 and x < 19950101);
//    }), D0.end());*/
//
//
//    //does not work this way for cuda 8, this style preferred till cuda 7.5
///*    struct test_functor
//    {
//        __host__ __device__
//        bool operator()(const int x){
//            return !(x >= 19940101 and x < 19950101);
//        }
//    };*/
//
//
//    // perform the filter operation and store result in another device vector
//    thrust::device_vector<int> device_result(D0.size());
//    auto result_end = thrust::copy_if(D0.begin(), D0.end(), device_result.begin(), [=] __device__(const int x) {
//        return (x >= 19940101 and x < 19950101);
//    });
//
//
//    // transfer the result back to host memory
//    thrust::host_vector<int> host_result(device_result.begin(), result_end);
//    std::cout << "result: " << host_result.size() << std::endl;
//
//    // complete implementation of find_if -> returns the first element found with criteria
///*    thrust::device_vector<int>::iterator iter;
//    iter = thrust::find_if(thrust::device, D0.begin(), D0.end(), [=] __device__(const int x) {
//        return !(x >= 19940101 and x < 19950101);
//    });*/}
//
////***************************************************************************************************//
////******************************* MAIN FUNCTION *****************************************************//
////***************************************************************************************************//
//
//int main(void)
//{
////    get the table content as 2D vector
//    std::vector<std::vector<int>> lineitemData;
//    lineitemData = getLineItemData();
//
//    // transpose the table data to read in column format
//    std::vector<std::vector<int>> transposedVec = getTransposedVector(lineitemData);
//
//    // send the column data necessary for the query - this hardcoding will be replaced by parser
//    std::vector<int> column_shipdate = transposedVec[8];
//    std::vector<int> column_discount = transposedVec[6];
//    std::vector<int> column_quantity = transposedVec[4];
//
//    af::array result1,result2,result3,result4,result5;
//    af::array temp;
//
//    auto start = high_resolution_clock::now();
//
//    result1 = filter_arrayfire(column_shipdate,"GE",19940101);
//    // write a buffer logic to check if the column data is already in device memory
//    // the logic could be to hold 'n' column data slots in map at a time
//    // or calculate the size of column data using sizeof(vec) and extend the map such that it is lesser than device memory
//    // when one of the case invalidates, evict the column data
//    result2 = filter_arrayfire(column_shipdate,"LE",19950101);
//    result3 = filter_arrayfire(column_discount,"GE",5);
//    result4 = filter_arrayfire(column_discount,"LE",7);
//    result5 = filter_arrayfire(column_quantity,"LT",24);
//
//    auto stop = high_resolution_clock::now();
//    auto duration = duration_cast<microseconds>(stop - start);
//
//    std::cout << "time taken for all predicates: " <<duration.count() << " microseconds" << std::endl;
//
//    start = high_resolution_clock::now();
//    temp = join_arrayfire(result1,result2);
//    temp = join_arrayfire(temp,result3);
//    temp = join_arrayfire(temp,result4);
//    temp = join_arrayfire(temp,result5);
//    stop = high_resolution_clock::now();
//    duration = duration_cast<microseconds>(stop - start);
//
//    std::cout << "time taken for all predicate conjunctions: " <<duration.count() << " microseconds" << std::endl;
//
////    af::print("result", temp);
//
//    return 0;
//}
//

#include "type_conversion/host_to_device.h"
#include "predicate/predicate.h"

int main(void) {

    //get the table content as 2D vector
    std::vector<std::vector<int>> lineitemData;
    lineitemData = getLineItemData();

    // transpose the table data to read in column format
    std::vector<std::vector<int>> transposedVec = getTransposedVector(lineitemData);

    std::vector<int> columnData = transposedVec[8];
    std::vector<int> durations_thr;
    for (int i = 0; i <= 5; i++) {
        auto start_thr = high_resolution_clock::now();
        thrust::device_vector<int> thrust_device_data = getThrustDeviceVector(columnData);
        auto stop_thr = high_resolution_clock::now();
        auto duration_thr = duration_cast<microseconds>(stop_thr - start_thr);
        durations_thr.push_back(duration_thr.count());
    }
    std::cout << "transfer time - Thrust: " << std::accumulate(durations_thr.begin(),
                                                               durations_thr.end(), 0) / durations_thr.size()
              << " microseconds" << std::endl;

    std::vector<int> durations_af;

    for (int i = 0; i <= 5; i++) {
        auto start_af = high_resolution_clock::now();
        af::array af_device_data = getAFDeviceVector(columnData);
        auto stop_af = high_resolution_clock::now();
        auto duration_af = duration_cast<microseconds>(stop_af - start_af);
        durations_af.push_back(duration_af.count());
    }
    std::cout << "transfer time - AF: " << std::accumulate(durations_af.begin(),
                                                           durations_af.end(), 0) / durations_af.size()
              << " microseconds" << std::endl;


    thrust::device_vector<int> thrust_device_data = getThrustDeviceVector(columnData);

    auto start_pred_thr = high_resolution_clock::now();
    predicate_thrust(thrust_device_data,19940101,"GE");
    auto stop_pred_thr = high_resolution_clock::now();
    auto duration_predthr = duration_cast<microseconds>(stop_pred_thr - start_pred_thr);
    std::cout << "Predicate - Thrust: " << duration_predthr.count()
              << " microseconds" << std::endl;

    af::array af_device_data = getAFDeviceVector(columnData);

    auto start_pred_af = high_resolution_clock::now();
    predicate_arrayfire(af_device_data,19940101,"GE");
    auto stop_pred_af = high_resolution_clock::now();
    auto duration_predAF = duration_cast<microseconds>(stop_pred_af - start_pred_af);
    std::cout << "Predicate - AF: " << duration_predAF.count()
              << " microseconds" << std::endl;

    return 0;
}
