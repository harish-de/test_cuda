//
// Created by harish on 17.07.20.
//
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>

#include <arrayfire.h>
#include <af/array.h>


#include <iostream>
#include <fstream>
#include <sstream>
#include<algorithm>

// forward declarations
std::vector<std::vector<int>> getLineItemData();
void filter_thrust();

std::vector<std::vector<int>> getTransposedVector(const std::vector<std::vector<int>> &lineitemData);

void testBackend()
{
    af::info();
    af_print(af::randu(5, 4));
}

void testArrayFire()
{
    try {
        printf("Trying CPU Backend\n");
        af::setBackend(AF_BACKEND_CPU);
        testBackend();
    } catch (af::exception& e) {
        printf("Caught exception when trying CPU backend\n");
        fprintf(stderr, "%s\n", e.what());
    }

    try {
        printf("Trying CUDA Backend\n");
        af::setBackend(AF_BACKEND_CUDA);
        testBackend();
    } catch (af::exception& e) {
        printf("Caught exception when trying CUDA backend\n");
        fprintf(stderr, "%s\n", e.what());
    }

    try {
        printf("Trying OpenCL Backend\n");
        af::setBackend(AF_BACKEND_OPENCL);
        testBackend();
    } catch (af::exception& e) {
        printf("Caught exception when trying OpenCL backend\n");
        fprintf(stderr, "%s\n", e.what());
    }


    // Create an array on the host, copy it into an ArrayFire 2x3 ArrayFire
    // array
    float host_ptr[] = {0, 1, 2, 3, 4, 5};
    af::array a(2, 3, host_ptr);

    // Create a CUDA device pointer, populate it with data from the host
    float *device_ptr;
    hipMalloc((void **)&device_ptr, 6 * sizeof(float));
    hipMemcpy(device_ptr, host_ptr, 6 * sizeof(float), hipMemcpyHostToDevice);

    // Convert the CUDA-allocated device memory into an ArrayFire array:
    af::array b(2, 3, device_ptr, afDevice);  // Note: afDevice (default: afHost)
    // Note that ArrayFire takes ownership over `device_ptr`, so memory will
    // be freed when `b` id destructed. Do not call hipFree(device_ptr)!
}

void filter_arrayfire(){

    // get the table content as 2D vector
    std::vector<std::vector<int>> lineitemData;
    lineitemData = getLineItemData();

    // transpose the table data to read in column format
    std::vector<std::vector<int>> transposedVec = getTransposedVector(lineitemData);

    std::cout << transposedVec.size() << std::endl;
    std::cout << transposedVec[8].size() << std::endl;

//    std::vector<int> temp = transposedVec[0];
    int* lineitem_date = &transposedVec[8][0]; //&temp[0];

    // copy host data to device
    af::array deviceDate((dim_t)6001215, lineitem_date);

//    af::array result = af::operator>>(deviceDate, 19940101);
    af::array index = af::where(af::operator>(deviceDate, 19940101));
    af::print("result", index);
}


int main(void)
{
//    filter_thrust();
    filter_arrayfire();
    return 0;
}

void filter_thrust() {// this following check confirmed addition of --expt-extended-lambda in CmakeLists.txt
// is necessary to executed lambda functions
#ifndef __CUDACC_EXTENDED_LAMBDA__
    std::cout << "compile with extended lambdas " << std::endl;
#endif

    // get the table content as 2D vector
    std::vector<std::vector<int>> lineitemData;
    lineitemData = getLineItemData();

    // transpose the table data to read in column format
    std::vector<std::vector<int>> transposedVec = getTransposedVector(lineitemData);

    // the table data in column format is host vector H
    thrust::host_vector<std::vector<int>> H(transposedVec);

    std::cout << "H has size " << H.size() << std::endl;
    std::cout << "H has size " << H[0].size() << std::endl;

    // the predicate column data is transferred/copied to the device vector
    thrust::device_vector<int> D0 = H[8];
    std::cout << "D has size before filtering " << D0.size() << std::endl;

    // Remove_If operation from thrust on device vector
/*    D0.erase(thrust::remove_if(D0.begin(), D0.end(), [=] __device__(const int x){
        return !(x >= 19940101 and x < 19950101);
    }), D0.end());*/


    //does not work this way for cuda 8, this style preferred till cuda 7.5
/*    struct test_functor
    {
        __host__ __device__
        bool operator()(const int x){
            return !(x >= 19940101 and x < 19950101);
        }
    };*/


    // perform the filter operation and store result in another device vector
    thrust::device_vector<int> device_result(D0.size());
    auto result_end = thrust::copy_if(D0.begin(), D0.end(), device_result.begin(), [=] __device__(const int x) {
        return (x >= 19940101 and x < 19950101);
    });

    // transfer the result back to host memory
    thrust::host_vector<int> host_result(device_result.begin(), result_end);
    std::cout << "result: " << host_result.size() << std::endl;

    // complete implementation of find_if -> returns the first element found with criteria
/*    thrust::device_vector<int>::iterator iter;
    iter = thrust::find_if(thrust::device, D0.begin(), D0.end(), [=] __device__(const int x) {
        return !(x >= 19940101 and x < 19950101);
    });*/}

std::vector<std::vector<int>> getTransposedVector(const std::vector<std::vector<int>> &lineitemData) {
    std::vector<std::vector<int>> transposedVec(lineitemData[0].size(),
                                                std::vector<int>(lineitemData.size()));
    for (size_t i = 0; i < lineitemData.size(); ++i)
        for (size_t j = 0; j < lineitemData[0].size(); ++j)
            transposedVec[j][i] = lineitemData[i][j];
    return transposedVec;
}


std::vector<std::vector<int>> getLineItemData() {
    std::string filename = "/home/harish/Documents/lineitem/lineitem.tbl";

    std::vector<std::vector<int>> lineitemData;

    std::vector<int> rowVector;

    std::ifstream lineitemFile(filename);

/*    if(!lineitemFile.is_open())
        throw std::runtime_error("could not open file");*/

    std::string row;
    std::string values;

    while (lineitemFile.good()) {
        std::getline(lineitemFile, row);
        std::stringstream ss(row);
        int count = 0;
        while (std::getline(ss,values,'|')){
            if(count < 8 or (count >=10 and count <=12)){
                if(count >= 4 and count <=7) {
                    int val = std::stof(values) * 100;
                    rowVector.push_back(val);
                }
                else if(count >=10 and count <=12){
                    values.erase(remove(values.begin(), values.end(), '-'), values.end());
                    rowVector.push_back(std::stoi(values));
                }
                else{
                    rowVector.push_back(std::stoi(values));
                }
            }
            count++;
        }
        lineitemData.push_back(rowVector);
        rowVector.clear();
    }
    return lineitemData;
}




