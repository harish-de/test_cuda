#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>


#include <iostream>
#include <fstream>
#include <sstream>
#include<algorithm>

#define DSIZE 10

int main(void)
{

#ifndef __CUDACC_EXTENDED_LAMBDA__
    std::cout << "compile with extended lambdas " << std::endl;
#endif

    std::string filename = "/home/harish/Documents/lineitem/lineitem.tbl";

    std::vector<std::vector<int>> lineitemData;

    std::vector<int> rowVector;

    std::ifstream lineitemFile(filename);

    if(!lineitemFile.is_open())
        throw std::runtime_error("could not open file");

    std::string row;
    std::string values;

    while (lineitemFile.good()) {
        std::getline(lineitemFile, row);
        std::stringstream ss(row);
        int count = 0;
        while (std::getline(ss,values,'|')){
            if(count < 8 or (count >=10 and count <=12)){
                if(count >= 4 and count <=7) {
                    int val = std::stof(values) * 100;
                    rowVector.push_back(val);
                }
                else if(count >=10 and count <=12){
                    values.erase(remove(values.begin(), values.end(), '-'), values.end());
                    rowVector.push_back(std::stoi(values));
                }
                else{
                    rowVector.push_back(std::stoi(values));
                }
            }
            count++;
        }
        lineitemData.push_back(rowVector);
        rowVector.clear();
    }

    std::vector<std::vector<int>> transposedVec(lineitemData[0].size(),
                                    std::vector<int>(lineitemData.size()));
    for (size_t i = 0; i < lineitemData.size(); ++i)
        for (size_t j = 0; j < lineitemData[0].size(); ++j)
            transposedVec[j][i] = lineitemData[i][j];

    thrust::host_vector<std::vector<int>> H(transposedVec);

    std::cout << "H has size " << H.size() << std::endl;
    std::cout << "H has size " << H[0].size() << std::endl;

    thrust::device_vector<int> D0 = H[8];
    std::cout << "D has size before filtering " << D0.size() << std::endl;

    // Remove_If operation from thrust on device vector
/*    D0.erase(thrust::remove_if(D0.begin(), D0.end(), [=] __device__(const int x){
        return !(x >= 19940101 and x < 19950101);
    }), D0.end());*/

/* does not work this way for cuda 8, this style preferred till cuda 7.5
 * struct test_functor
    {
        __host__ __device__
        bool operator()(const int x){
            return !(x >= 19940101 and x < 19950101);
        }
    };*/

    thrust::device_vector<int> device_result(D0.size());
    auto result_end = thrust::copy_if(D0.begin(), D0.end(), device_result.begin(), [=] __device__(const int x) {
        return (x >= 19940101 and x < 19950101);
    });

    thrust::host_vector<int> host_result(device_result.begin(), result_end);
    std::cout << "result: " << host_result.size() << std::endl;

/* complete implementation of find_if -> returns the first element found with criteria
 * thrust::device_vector<int>::iterator iter;
    iter = thrust::find_if(thrust::device, D0.begin(), D0.end(), [=] __device__(const int x) {
        return !(x >= 19940101 and x < 19950101);
    });*/
    return 0;
}




/*
#include <arrayfire.h>
#include <af/array.h>

void testBackend()
{
    af::info();
    af_print(af::randu(5, 4));
}

int main()
{
    */
/*try {
        printf("Trying CPU Backend\n");
        af::setBackend(AF_BACKEND_CPU);
        testBackend();
    } catch (af::exception& e) {
//        printf("Caught exception when trying CPU backend\n");
        fprintf(stderr, "%s\n", e.what());
    }

    try {
        printf("Trying CUDA Backend\n");
        af::setBackend(AF_BACKEND_CUDA);
        testBackend();
    } catch (af::exception& e) {
        printf("Caught exception when trying CUDA backend\n");
        fprintf(stderr, "%s\n", e.what());
    }

    try {
        printf("Trying OpenCL Backend\n");
        af::setBackend(AF_BACKEND_OPENCL);
        testBackend();
    } catch (af::exception& e) {
        printf("Caught exception when trying OpenCL backend\n");
        fprintf(stderr, "%s\n", e.what());
    }*//*


    // Create an array on the host, copy it into an ArrayFire 2x3 ArrayFire
    // array
    float host_ptr[] = {0, 1, 2, 3, 4, 5};
    af::array a(2, 3, host_ptr);

    // Create a CUDA device pointer, populate it with data from the host
    float *device_ptr;
    hipMalloc((void **)&device_ptr, 6 * sizeof(float));
    hipMemcpy(device_ptr, host_ptr, 6 * sizeof(float), hipMemcpyHostToDevice);

    // Convert the CUDA-allocated device memory into an ArrayFire array:
    af::array b(2, 3, device_ptr, afDevice);  // Note: afDevice (default: afHost)
    // Note that ArrayFire takes ownership over `device_ptr`, so memory will
    // be freed when `b` id destructed. Do not call hipFree(device_ptr)!

    return 0;
}*/
